#include "hip/hip_runtime.h"
#include <cstdio>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/fill.h>
#include <nuke/atomic.hpp>

__global__ void inc_kernel(nuke::atomic_int *a_ptr)
{
  (*a_ptr)++;
}

__global__ void dec_kernel(nuke::atomic_int *a_ptr)
{
  (*a_ptr)--;
}

__global__ void print_kernel(nuke::atomic_int *a_ptr)
{
  printf("result is %d\n", (int)*a_ptr);
}

int main()
{
  thrust::device_ptr<nuke::atomic_int> a_ptr = thrust::device_malloc<nuke::atomic_int>(1);

  thrust::fill(a_ptr, a_ptr + 1, 0);

  inc_kernel<<<2,512>>>(a_ptr.get());

  print_kernel<<<1,1>>>(a_ptr.get());

  dec_kernel<<<2,512>>>(a_ptr.get());

  print_kernel<<<1,1>>>(a_ptr.get());

  thrust::device_free(a_ptr);

  return 0;
}

